#include "hip/hip_runtime.h"
#include "cuda_math.h"
#include <stdio.h>
#include <stdlib.h>
#include "structs.cuh"

#include "im2D.h"
#include "im3D.hpp"

#include "LBMconsts.cuh"

#include "phys.h"

im3D_pars im3DHost;
void calcStep(int REV=1);

const char* FuncStr[] = {
  "rho", "Vx", "Vy", "Vz",
  "f0","f1","f2","f3","f4"
};

__global__ void __launch_bounds__(Nz) draw(float* buf) {
  int iz=threadIdx.x;
  int ix=blockIdx.x;
  int iy=blockIdx.y;

  float* pbuf=&buf[ix+gridDim.x*(iy+gridDim.y*iz)];
  register Cell cell = pars.data.get_cell_compact<0>(ix,iy,iz);
  cell.updateRhoVel();
  ftype rho=0; rho=cell.rho;
  ftype3 vel=make_ftype3(0,0,0);
  vel = cell.vel;
  
  switch(pars.nFunc) {
      case 0 : *pbuf=float(rho); break;
      case 1 : *pbuf=float(vel.x); break;
      case 2 : *pbuf=float(vel.y); break;
      case 3 : *pbuf=float(vel.z); break;
      case 4 : *pbuf=float(cell.f[0]); break;
      case 5 : *pbuf=float(cell.f[1]); break;
      case 6 : *pbuf=float(cell.f[2]); break;
      case 7 : *pbuf=float(cell.f[3]); break;
      case 8 : *pbuf=float(cell.f[4]); break;
      default: break;
  }
}
void draw_all(){
  CHECK_ERROR( hipMemset(parsHost.arr4im.Arr3Dbuf,0,((long long int)parsHost.arr4im.Nx)*parsHost.arr4im.Ny*parsHost.arr4im.Nz*sizeof(float)) );
  draw<<<dim3(parsHost.arr4im.Nx,parsHost.arr4im.Ny),parsHost.arr4im.Nz>>>(parsHost.arr4im.Arr3Dbuf);
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  im3DHost.initCuda(parsHost.arr4im);
}

void idle_func_calc::step() {
  for(int ii=0;ii<PPhost.StepIterPeriod;ii++) {
    calcStep(); 
    t++;
  }
  draw_all();
  im3DHost.save_png(parsHost.iStep/PPhost.StepIterPeriod);
  recalc_at_once=true;
}

static void key_func(unsigned char key, int x, int y) {
  if(type_diag_flag>=2) printf("keyN=%d, coors=(%d,%d)\n", key, x, y);
  if(key == 'h') {
    printf("\
======= Управление:\n\
  <¦>  \tИзменение функции для визуализации: Values_level543210¦isBnd_level543210¦AMR_LEVEL\n\
«Enter»\tПересчёт одного шага\n\
   b   \tвключает пересчёт в динамике (см. «Управление динамикой»)\n\
"); im3DHost.print_help();
    return;
  }
  ftype t0;
  switch(key) {
  //case '>': if(parsHost.nFunc<parsHost.MaxFunc) parsHost.nFunc++; break;
  //case '<': if(parsHost.nFunc>0) parsHost.nFunc--; break;
  case '>': parsHost.nFunc = (parsHost.nFunc+1)%parsHost.MaxFunc; break;
  case '<': parsHost.nFunc = (parsHost.nFunc+parsHost.MaxFunc-1)%parsHost.MaxFunc; break;
  case 13: for(int ii=0;ii<PPhost.StepIterPeriod;ii++) calcStep(+1); /*im3DHost.save_png(parsHost.iStep/PPhost.StepIterPeriod)*/; break;
  case 8 : for(int ii=0;ii<PPhost.StepIterPeriod;ii++) calcStep(-1); /*im3DHost.save_png(parsHost.iStep/PPhost.StepIterPeriod)*/; break;
  default: if(!im3DHost.key_func(key, x, y)) {
  if(type_diag_flag>=0) printf("По клавише %d в позиции (%d,%d) нет никакого действия\n", key, x, y);
  } return;
  }
  copy2dev( parsHost, pars );
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  draw_all();
  recalc_at_once=true;
}
static void draw_func() {
  im3DHost.fName = FuncStr[parsHost.nFunc]; 
  im2D.draw(im3DHost.reset_title()); 
}

static void idle_func() { im3DHost.recalc_func(); }
static void mouse_func(int button, int state, int x, int y) { im3DHost.mouse_func(button, state, x, y); }
static void motion_func(int x, int y) { im3DHost.motion_func(x, y); }
static void special_func(int key, int x, int y) { 
  im3DHost.special_func(key, x, y);
  if(key == GLUT_KEY_F2) {
//    parsHost.drawArrows^=1;
    copy2dev( parsHost, pars ); draw_all();
    recalc_at_once=true;
  }
}

int print_help() {
  printf("help | using in test|batch mode:\n ./lbm [--help|--test|--batch]\n");
  printf("using in interactive mode:\n ./lbm %s\n", im3DHost.command_line_help_string());
  im3DHost.print_command_line_help();
  return 0;
}
void read_float3(float* v, char* str);
float read_float(char* str);

void launch_im3D(int argc, char** argv){
  parsHost.nFunc = 0; parsHost.MaxFunc = sizeof(FuncStr)/sizeof(char*);
    
  cudaTimer tm; tm.start();
  parsHost.reset_im();
  im3DHost.reset(parsHost.arr4im);
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );
  im2D.get_device(3,0);
  im2D.init_image(argc,argv, im3DHost.bNx, im3DHost.bNy, "im3D");
  im3DHost.init3D(parsHost.arr4im); im3DHost.iz0=parsHost.arr4im.Nx-1; im3DHost.key_func('b',0,0);
  im3DHost.initCuda(parsHost.arr4im);
  draw_all();

  if(type_diag_flag>=1) printf("Настройка GLUT и запуск интерфейса\n");
  glutIdleFunc(idle_func);
  glutKeyboardFunc(key_func);
  glutMouseFunc(mouse_func);
  glutMotionFunc(motion_func);
  glutDisplayFunc(draw_func);
  glutSpecialFunc(special_func);
  if(type_diag_flag>=0) printf("Init cuda device: %.1f msec\n", tm.stop());
  glutMainLoop();
}

float get_val_from_arr3D(int ix, int iy, int iz) {
  Arr3D_pars& arr=parsHost.arr4im;
  if(arr.inCPUmem) return arr.Arr3Dbuf[arr.get_ind(ix,iy,iz)];
  float res=0.0;
  if(arr.inGPUmem) CHECK_ERROR(hipMemcpy(&res, arr.get_ptr(ix,iy,iz), sizeof(float), hipMemcpyDeviceToHost));
  return res;
}

