//#include <hip/hip_fp16.h>
#include "cuda_math.h"
//#include "cuda_math_double.h"
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <error.h>
#include "structs.cuh"
#include "init.h"

#include "im2D.h"
#include "im3D.hpp"

#include "phys.h"

int type_diag_flag=0;
extern im3D_pars im3DHost;
AllParamsHost parsHost;
__constant__ AllParams pars;

PhysPars PPhost;
__constant__ PhysPars PPdev;
void PhysPars::MallocData(){};
void PhysPars::setCell(int val, int x,int y){};

void AllParamsHost::set(){
  //PPhost.setUnits();
  sprintf(im3DHost.drop_dir, "%s", PPhost.drop_dir);
  Nt=PPhost.Nt;
  MaxSteps=PPhost.MaxSteps;

  //if(Nx%(1<<MaxLevel)!=0) error(1,1,"Error: Nx must be dividable by %d\n", 1<<MaxLevel);
}

int print_help();
void launch_im3D(int argc, char** argv);
bool interactive=true, test_only=false;
void reset(im3D_pars* p=0);
void init();
void simple_drop();
void calcStep(int REV=1);
int _main(int argc, char** argv) {
  int Ndevs=0; CHECK_ERROR( hipGetDeviceCount(&Ndevs) ); CudaDevs=Ndevs;
  CudaDevs=1;
  ::reset();
  argv ++; argc --;
  im3DHost.reset();
  while(argc>0 && strncmp(*argv,"--",2)==0) {
    int pp=1;
    if(strcmp(*argv,"--test")==0) test_only = true;
    else if(strcmp(*argv,"--batch")==0) interactive = false;
    else pp = im3DHost.init_from_command_line(argv);
    if(pp<=0) return print_help();
    else if(pp==1) printf("par: %s; \n", argv[0]);
    else if(pp==2) printf("par: %s; vals: %s\n", argv[0], argv[1]);
    argv += pp; argc -= pp;
  };
  if(test_only) printf("No GL\n");
  else printf("With GL\n");
  im2D.get_device(3,0);
  type_diag_flag = 1;
try {
  if(type_diag_flag>=1) printf("Настройка опций визуализации по умолчанию\n");
  cudaTimer tm; tm.start();
  parsHost.set();
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  init();
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );

  if(test_only) {
    while(parsHost.iStep<parsHost.MaxSteps) {
      tm.start();
      calcStep();
      double tCpu=tm.stop();
//      fprintf(parsHost.fLog,"run time: %.2f msec, %.2f MLU/sec\n", tCpu, 1.e-6*Nx*Ny*Nz/tCpu);
      printf("run time: %.2f msec, %.2f MLU/sec\n", tCpu, 1.e-6*Nx*Ny*Nz/tCpu);
    }
    return 0;
  } else{
    launch_im3D(argc,argv);
  }
} catch(...) {
  printf("Возникла какая-то ошибка.\n");
}
  parsHost.clear();
  return -1;
}
int main(int argc, char** argv) {
  PPhost.setDefault();
  return _main(argc,argv);
}
int run(int argc, char** argv) {
  return _main(argc,argv);
}
