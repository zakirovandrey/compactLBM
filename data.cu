#include "data.cuh"


__host__ __device__ void Cell::updateRhoVel(){
  using namespace LBMconsts;
  vel = make_ftype3(0,0,0); rho=0;
  for(int iq=0; iq<Qn; iq++) { rho+=f[iq]; vel+= make_ftype3(e[iq])*f[iq]; }
  if(rho!=0) vel/=rho;
}

void Data_t::malloc_data(const int Nx, const int Ny, const int Nz){
  const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
  const size_t sz = long(Nx)*Ny*Nz/Ns3*sizeof(Tile);
  printf("Total data size = %g GB\n",double(sz)/1024/1024/1024); 
  CHECK_ERROR( hipMalloc((void**)&tiles, sz ) );
  CHECK_ERROR( hipHostMalloc((void**)&tilesHost, sz ) );
  CHECK_ERROR( hipMemset(tiles, 0, sz ) );
  CHECK_ERROR( hipMemset(tilesHost, 0, sz ) );
};
void Data_t::copyHost2Dev(){
  const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
  const size_t sz = long(Nx)*Ny*Nz/Ns3*sizeof(Tile);
  CHECK_ERROR( hipMemcpy(tiles, tilesHost, sz, hipMemcpyHostToDevice ) );
}
void Data_t::copyDev2Host(){
  const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
  const size_t sz = long(Nx)*Ny*Nz/Ns3*sizeof(Tile);
  CHECK_ERROR( hipMemcpy(tiles, tilesHost, sz, hipMemcpyDeviceToHost ) );
}


