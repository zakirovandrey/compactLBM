#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "err.h"
namespace errors{
__managed__ hipError_t last_err=hipSuccess;
};

void PrintLastError(const char *file, int line) {
  hipError_t err=hipGetLastError();
  if(err!=hipSuccess) fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
}
bool CheckError(hipError_t err, const char *file, int line) {
  hipError_t dev_err; hipMemcpy(&dev_err, &errors::last_err, sizeof(hipError_t), hipMemcpyDefault);
  if(err==hipSuccess && dev_err==hipSuccess) return false;
  if(err==hipSuccess) err = errors::last_err;
  fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
  return true;
}
bool __device__ CheckErrorDevice(hipError_t err, const char *file, int line) {
  if(err==hipSuccess) return false;
  atomicCAS((int*)(&errors::last_err), hipSuccess, err);
  return true;
}

void deviceDiagnostics(){
  int deviceCount;
  CHECK_ERROR( hipGetDeviceCount(&deviceCount) );  
  printf("GPU devices :: %d \n", deviceCount);
  hipDeviceProp_t devProp[deviceCount];
  for(int i = 0; i < deviceCount; ++i) {
    printf("*** CUDA Device #%d ***", i);
    CHECK_ERROR( hipGetDeviceProperties(&devProp[i], i) );
    printf("%s ***\n", devProp[i].name);
    printf("\t%d.%d compute capability\n", devProp[i].major, devProp[i].minor);
    printf("\t%d multiprocessors\n", devProp[i].multiProcessorCount);
    printf("\t%.2fGB max mem pitch of %.2fGB global memory\n", devProp[i].memPitch/(1024.*1024.*1024), devProp[i].totalGlobalMem/(1024.*1024.*1024));
    printf("\t%.2fKB total shared memory per block\n", devProp[i].sharedMemPerBlock/1024.);
    printf("\t%.2fKB total constant memory\n", devProp[i].totalConstMem/1024.);
    printf("\t%.2fK registers per block\n", devProp[i].regsPerBlock/1024.);
    printf("\t%d/%d threads per Warp/block\n", devProp[i].warpSize, devProp[i].maxThreadsPerBlock);
    printf("\tClock rate: %.2fGHz\n", devProp[i].clockRate*1e-6);
    printf("\tTexture alignment: %luB\n", devProp[i].textureAlignment);
    printf("\tConcurrent copy and execution: %s\n", (devProp[i].deviceOverlap ? "Yes" : "No"));
    printf("\tKernel execution timeout: %s\n", (devProp[i].kernelExecTimeoutEnabled ? "Yes" : "No"));
  }
}
#include "im3D.hpp"
extern bool recalc_always; 
int im3D_pars4save::init_from_command_line(char** argv) {
  if(strncmp(*argv,"--help",6)==0) return -1;
  if(strncmp(*argv,"--devQ",6)==0) { deviceDiagnostics(); return 1; }
  if(strcmp(*argv,"--box")==0) read_float3(BoxFactor, argv[1]);
  else if(strcmp(*argv,"--load")==0) load_from_file(argv[1]);
  else if(strcmp(*argv,"--mesh")==0) read_float3(MeshBox, argv[1]);
  else if(strcmp(*argv,"--sh_mesh")==0) read_float3(MeshShift, argv[1]);
  else if(strcmp(*argv,"--Dmesh")==0) Dmesh=read_float(argv[1]);
  else if(strcmp(*argv,"--zoom")==0) read_float3(Dzoom, argv[1]);
  else if(strcmp(*argv,"--add")==0) read_float3(Dadd, argv[1]);
  else if(strcmp(*argv,"--shrink")==0) read_int3(Dshrink, argv[1]);
  else if(strcmp(*argv,"--Narr")==0) read_int3(Narr, argv[1]);
  else if(strcmp(*argv,"--step")==0) read_float3(step, argv[1]);
  else if(strcmp(*argv,"--base")==0) read_float3(base, argv[1]);
  else if(strcmp(*argv,"--bkgr_col")==0) read_float3(bkgr_col, argv[1]);
  else if(strcmp(*argv,"--mesh_col")==0) read_float3(mesh_col, argv[1]);
  else if(strcmp(*argv,"--box_col")==0) read_float3(box_col, argv[1]);
  else if(strcmp(*argv,"--rot_point")==0) read_float3(RotPoint, argv[1]);
  else if(strcmp(*argv,"--box_shrink")==0) read_float3(box_shrink, argv[1]);
  else if(strcmp(*argv,"--drop_dir")==0) strcpy(drop_dir,argv[1]);
  else if(strcmp(*argv,"--cntr")==0) cntr_levels[cntr_num++]=read_float(argv[1]);
  else if(strcmp(*argv,"--ld_sz")==0) read_int2(ld_sz, argv[1]);
  else if(strcmp(*argv,"--recalc_always")==0) { recalc_always=true; return 1; }
  else if(strcmp(*argv,"--cntr_clear")==0) { cntr_num=0; return 1; }
  else if(strcmp(*argv,"--nocomp")==0) return 1;
  else if(strcmp(*argv,"--norun")==0) return 1;
  else if(strcmp(*argv,"--redefine")==0) return 2;
  else { printf("Illegal parameters' syntax notation\n<%s>", *argv); return 0; }
  //else if(strcmp(*argv,"--")==0) read_float3(, argv[1]);
  //printf("par: %s; vals: %s\n", argv[0], argv[1]);
  return 2;
}
const char* im3D_pars4save::command_line_help_string() {
  return "[--devQ] [--load <opt-file>] [--zoom \"1. 1. 1.\"] [--shrink \"1 1 1\"] [--step \"1. 1. 1.\"] [--base \"1. 1. 1.\"] [--box \"1. 1. 1.\"] [--mesh \"200. 200. 200.\"] [--Dmesh 5.] [--drop_dir \".\"] [--bkgr_col \"0.1 0.1 0.1\"] [--mesh_col \"0.8 0.8 0.2\"] [--box_col \"1. 1. 1.\"] [--box_shrink \"1. 1. 1.\"] [--sensor \"1 1 1\"]";
}
void im3D_pars4save::print_command_line_help() {
  printf("  --devQ\tВыдаёт информацию о видеокартах на компьютере;\n");
  printf("  --load\tВводит параметры из файла <opt-file>, сохранённые ранее клавишей <w/W>\n");
  printf("  --zoom\tмасштабный фактор, действует на 2D режим и размер окна, [1. 1. 1.];\n");
  printf("  --add \tдобавляет пространство к размеру окна. Требуется для вывода 3D на фоне 2D, [0. 0. 0.];\n");
  printf("  --shrink\tмасштабный фактор, действует везде, сокращает требования к памяти, [1 1 1];\n");
  printf("  --Narr\tявно заданный размер массива (если =0, берётся из первого файла) [0 0 0];\n");
  printf("  --box \tкоррекция пропорций размера бокса в 3D режиме, [1. 1. 1.];\n");
  printf("  --step \tшаги между точками, действует только на тики, [1. 1. 1.];\n");
  printf("  --base \tшаги между точками, действует только на тики, [0. 0. 0.];\n");
  printf("  --mesh\tрасстояние между линиями сетки в боксе по координатам в ячейках (до коррекции), [100. 100. 100.];\n");
  printf("  --sh_mesh\tсдвиг линий сетки в боксе по координатам в ячейках (до коррекции), [0. 0. 0.];\n");
  printf("  --Dmesh\tширина линии сетки в пикселях (со сглаживанием выглядит несколько уже), [5.];\n");
  printf("  --drop_dir\tимя директории, в которую будут сохраняться различные файлы, [.];\n");
  printf("  --bkgr_col\tцвет фона, [0.1 0.1 0.1];\n");
  printf("  --mesh_col\tцвет линий сетки, [0.8 0.8 0.2];\n");
  printf("  --box_col\tцвет линий бокса, [1.0 1.0 1.0];\n");
  printf("  --box_shrink\t коэффициент растяжения размеров бокса, [1.0 1.0 1.0];\n");
  printf("  --rot_point\t точка в боксе, относительно которой производится вращение, [0.5 0.5 0.5];\n");
  printf("  --sensor\tкоординаты сенсора, можно задавать несколько сенсоров;\n");
  printf("  --cntr\tзначение уровня контура, можно задавать несколько уровней;\n");
  printf("  --cntr_clear\tочищает все ранее заданные значения уровней контура;\n");
  printf("  --ld_sz\tчтение сохраненных ранее параметров в режиме совместимости, [80 288];\n");
}
